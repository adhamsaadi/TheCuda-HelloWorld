#include "hip/hip_runtime.h"
#include <stdio.h>

const int N = 13; // Number of elements in the arrays
const int blocksize = 6; // Number of threads per block

__global__ 
void hello(char* a, int* b) 
{
    // Ensure threads only access valid indices
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < N) {
        a[idx] += b[idx];
    }
}

int main()
{
    char a[N] = "Hello \0\0\0\0\0\0";
    int b[N] = {15, 10, 6, 0, -11, 1, 0, 0, 0, 0, 0, 0, 0};

    char* device_a;
    int* device_b;
    const int csize = N * sizeof(char);
    const int isize = N * sizeof(int);

    // Print the initial string
    printf("%s\n", a);

    // Allocate memory on the device
    hipMalloc((void**)&device_a, csize);
    hipMalloc((void**)&device_b, isize);

    // Copy data from host to device
    hipMemcpy(device_a, a, csize, hipMemcpyHostToDevice);
    hipMemcpy(device_b, b, isize, hipMemcpyHostToDevice);

    // Configure the grid and block dimensions
    dim3 dimBlock(min(N, blocksize));
    dim3 dimGrid((N + blocksize - 1) / blocksize);

    // Launch the kernel
    hello<<<dimGrid, dimBlock>>>(device_a, device_b);

    // Copy the result back to the host
    hipMemcpy(a, device_a, csize, hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(device_a);
    hipFree(device_b);

    // Print the modified string
    printf("%s\n", a);

    return 0;
}
